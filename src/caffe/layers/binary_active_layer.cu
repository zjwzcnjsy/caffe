#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/binary_active_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void BinaryActiveForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = sign(in[index]);
  }
}

template <typename Dtype>
void BinaryActiveLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  BinaryActiveForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void BinaryActiveBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * (in_data[index] > Dtype(-1.) && in_data[index] < Dtype(1.));
  }
}

template <typename Dtype>
void BinaryActiveLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    BinaryActiveBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(BinaryActiveLayer);


}  // namespace caffe
