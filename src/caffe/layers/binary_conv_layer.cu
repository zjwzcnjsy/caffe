#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/binary_conv_layer.hpp"
//#include "hip/hip_runtime_api.h"
//#include "caffe/util/benchmark.hpp"

namespace caffe {

	template <typename Dtype>
	__global__ void binarize(const int nThreads, const int kernel_dim,
		const Dtype* w, const Dtype* A, Dtype* w_bin, Dtype* w_sign) {
		CUDA_KERNEL_LOOP(index, nThreads) {
			int num = index / kernel_dim;
			Dtype v = sign(w[index]);
			w_sign[index] = v;
			w_bin[index] = v * A[num];
		}
	}

	template <typename Dtype>
	__global__ void permute_channel(const int nThreads,
		const int num, const int channels, const int height, const int width,
		const Dtype* weights, Dtype* buffer) {
		CUDA_KERNEL_LOOP(index, nThreads) {
			const int w = index % width;
			const int h = (index / width) % height;
			const int c = (index / width / height) % channels;
			const int n = index / width / height / channels;
			buffer[((n*height + h)*width + w)*channels + c] = weights[index];
		}
	}

	template <typename Dtype>
	__global__ void calc_meancenter(const int nThreads,
		const int num, const int channels, const int height, const int width,
		const Dtype* weights, Dtype* center) {
		CUDA_KERNEL_LOOP(index, nThreads) {
			const int w = index % width;
			const int h = (index / width) % height;
			const int n = index / width / height;
			Dtype sum = Dtype(0.0);
			for (int c = 0; c < channels; ++c) {
				sum += weights[((n*channels + c)*height + h)*width + w];
			}
			center[(n*height + h)*width + w] = sum / static_cast<Dtype>(channels);
		}
	}

	template <typename Dtype>
	__global__ void meancenter_remove_and_clamp(const int nThreads,
		const int num, const int channels, const int height, const int width,
		const Dtype* center, Dtype* weights, const Dtype minv, const Dtype maxv) {
		CUDA_KERNEL_LOOP(index, nThreads) {
			const int w = index % width;
			const int h = (index / width) % height;
			const int n = index / width / height / channels;
			Dtype v = weights[index] - center[(n*height + h)*width + w];
			if (v < minv) {
				v = minv;
			}
			else if (v > maxv) {
				v = maxv;
			}
			else {
				//nothing to do;
			}
			weights[index] = v;
		}
	}

	template <typename Dtype>
	__global__ void mc_remove_clamp_abs(const int nThreads,
		const int num, const int channels, const int height, const int width,
		const Dtype* center, Dtype* weights, Dtype* abs_w, const Dtype minv, const Dtype maxv) {
		CUDA_KERNEL_LOOP(index, nThreads) {
			const int w = index % width;
			const int h = (index / width) % height;
			const int n = index / width / height / channels;
			Dtype v = weights[index] - center[(n*height + h)*width + w];
			if (v < minv) {
				v = minv;
			}
			else if (v > maxv) {
				v = maxv;
			}
			else {
				//nothing to do;
			}
			weights[index] = v;
			abs_w[index] = std::abs(v);
		}
	}

	template <typename Dtype>
	__global__ void meancenter_remove(const int nThreads,
		const int num, const int channels, const int height, const int width,
		const Dtype* center, Dtype* weights) {
		CUDA_KERNEL_LOOP(index, nThreads) {
			const int w = index % width;
			const int h = (index / width) % height;
			const int n = index / width / height / channels;
			weights[index] -= center[(n*height + h)*width + w];
		}
	}

	template <typename Dtype>
	__global__ void clamp(const int nThreads, Dtype* weights, const Dtype minv, const Dtype maxv) {
		CUDA_KERNEL_LOOP(index, nThreads) {
			Dtype v = weights[index];
			if (v < minv) {
				v = minv;
			}
			else if (v > maxv) {
				v = maxv;
			}
			else {
				//nothing to do;
			}
			weights[index] = v;
		}
	}

	template <typename Dtype>
	void BinaryConvolutionLayer<Dtype>::binarizeGPUTo(Blob<Dtype>* weights) {
		CHECK_EQ(weights->count(), binary_w_.count());
		CHECK_EQ(weights->num(), A_.num());
		const int count = weights->count();
		const int num = weights->num();
		const int kernel_dim = weights->count(1);
		// compute A
		//caffe_gpu_abs(count, weights->gpu_data(), w_buffer_.mutable_gpu_diff());
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num,
			1, kernel_dim, Dtype(1. / kernel_dim), w_buffer_.gpu_diff(), multiplier_.gpu_data(),
			(Dtype)0., A_.mutable_gpu_data());
		// compute sign(w) and A*sign(w)
		binarize<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
			count, kernel_dim, weights->gpu_data(), A_.gpu_data(), weights->mutable_gpu_data(), binary_w_.mutable_gpu_diff());
	}

	template <typename Dtype>
	void BinaryConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		//hipProfilerStart();
		if (this->phase_ == TRAIN) {
			const int count = this->blobs_[0]->count();
			const int num = this->blobs_[0]->num();
			const int channels = this->blobs_[0]->channels();
			const int height = this->blobs_[0]->height();
			const int width = this->blobs_[0]->width();
			// compute mean
			if (height == 1 && width == 1) {
				caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num,
					1, channels, Dtype(1. / channels), this->blobs_[0]->gpu_data(), multiplier_.gpu_data(),
					(Dtype)0., meancenter_.mutable_gpu_data());
			}
			else {
				permute_channel<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> > (
					count, num, channels, height, width,
					this->blobs_[0]->gpu_data(), w_buffer_.mutable_gpu_data());
				caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num*height*width,
					1, channels, Dtype(1. / channels), w_buffer_.gpu_data(), multiplier_.gpu_data(),
					(Dtype)0., meancenter_.mutable_gpu_data());
				/*calc_meancenter<Dtype> << <CAFFE_GET_BLOCKS(count / channels), CAFFE_CUDA_NUM_THREADS >> >(
					count / channels, num, channels, height, width,
					this->blobs_[0]->gpu_data(), meancenter_.mutable_gpu_data());*/
			}

			// subtract mean and clip weight to [-1,1]
			mc_remove_clamp_abs<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
				count, num, channels, height, width,
				meancenter_.gpu_data(), this->blobs_[0]->mutable_gpu_data(), w_buffer_.mutable_gpu_diff(), 
				Dtype(-1.0), Dtype(1.0));
			
			// store weight to buffer
			caffe_copy(count, this->blobs_[0]->gpu_data(), w_buffer_.mutable_gpu_data());

			// binary weight, this->blobs_[0]'s data hold A*sign(w), binary_w_'s diff hold sign(w)
			binarizeGPUTo(&(*this->blobs_[0]));
		}
		else {
			const int count = this->blobs_[0]->count();
			// store weight to buffer
			caffe_copy(count, this->blobs_[0]->gpu_data(), w_buffer_.mutable_gpu_data());

			const int num = this->blobs_[0]->num();
			const int channels = this->blobs_[0]->channels();
			const int height = this->blobs_[0]->height();
			const int width = this->blobs_[0]->width();
			// compute mean
			calc_meancenter<Dtype> << <CAFFE_GET_BLOCKS(count / channels), CAFFE_CUDA_NUM_THREADS >> >(
				count / channels, num, channels, height, width,
				this->blobs_[0]->gpu_data(), meancenter_.mutable_gpu_data());
			// subtract mean and clip weight to [-1,1]
			mc_remove_clamp_abs<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
				count, num, channels, height, width,
				meancenter_.gpu_data(), this->blobs_[0]->mutable_gpu_data(), w_buffer_.mutable_gpu_diff(),
				Dtype(-1.0), Dtype(1.0));
			// binary weight, binary_w_'s data hold A*sign(w), binary_w_'s diff hold sign(w)
			binarizeGPUTo(&(*this->blobs_[0]));
		}

		const Dtype* weight = this->blobs_[0]->gpu_data();
		for (int i = 0; i < bottom.size(); ++i) {
			const Dtype* bottom_data = bottom[i]->gpu_data();
			Dtype* top_data = top[i]->mutable_gpu_data();
			for (int n = 0; n < this->num_; ++n) {
				this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
					top_data + n * this->top_dim_);
				if (this->bias_term_) {
					const Dtype* bias = this->blobs_[1]->gpu_data();
					this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
				}
			}
		}
		if (this->phase_ == TEST) {
			const int count = this->blobs_[0]->count();
			// restore weight
			caffe_copy(count, w_buffer_.gpu_data(), this->blobs_[0]->mutable_gpu_data());
		}
		//hipProfilerStop();
	}

	template <typename Dtype>
	__global__ void A_backwark_kernel(const int nThreads, const int kernel_dim,
		const Dtype* w_sign, const Dtype* w_hat_grad, Dtype* A_grad) {
		CUDA_KERNEL_LOOP(index, nThreads) {
			Dtype sum = Dtype(0.);
			for (int i = 0; i < kernel_dim; ++i) {
				sum += w_sign[index*kernel_dim + i] * w_hat_grad[index*kernel_dim + i];
			}
			A_grad[index] = sum / static_cast<Dtype>(kernel_dim);
		}
	}

	template <typename Dtype>
	__global__ void meancenter_backwark_kernel(const int nThreads, const int num, const int channels, const int height, const int width,
		const Dtype* w_hat_grad, Dtype* meancenter_grad) {
		CUDA_KERNEL_LOOP(index, nThreads) {
			const int w = index % width;
			const int h = (index / width) % height;
			const int n = index / width / height;
			Dtype sum = Dtype(0.0);
			for (int c = 0; c < channels; ++c) {
				sum += w_hat_grad[((n*channels + c)*height + h)*width + w];
			}
			meancenter_grad[(n*height + h)*width + w] = sum / static_cast<Dtype>(channels);
		}
	}

	template <typename Dtype>
	__global__ void w_backwark_kernel(const int nThreads, const int num, const int channels, const int height, const int width,
		const Dtype* w, const Dtype* w_sign, const Dtype* w_hat_grad,
		const Dtype* A, const Dtype* A_grad, const Dtype* meancenter_grad, Dtype* w_grad) {
		CUDA_KERNEL_LOOP(index, nThreads) {
			const int w1 = index % width;
			const int h = (index / width) % height;
			const int n = index / width / height / channels;
			w_grad[index] = (w_sign[index] * A_grad[n] + w_hat_grad[index] * A[n] * (w[index] <= Dtype(1.) && w[index] >= Dtype(-1.)))
				* (Dtype(1.) - meancenter_grad[(n*height+h)*width+w1]);
		}
	}

	template <typename Dtype>
	void BinaryConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
		const Dtype* weight = this->blobs_[0]->gpu_data();
		Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
		for (int i = 0; i < top.size(); ++i) {
			const Dtype* top_diff = top[i]->gpu_diff();
			// Bias gradient, if necessary.
			if (this->bias_term_ && this->param_propagate_down_[1]) {
				Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
				for (int n = 0; n < this->num_; ++n) {
					this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
				}
			}
			if (this->param_propagate_down_[0] || propagate_down[i]) {
				const Dtype* bottom_data = bottom[i]->gpu_data();
				Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
				for (int n = 0; n < this->num_; ++n) {
					// gradient w.r.t. weight. Note that we will accumulate diffs.
					if (this->param_propagate_down_[0]) {
						this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
							top_diff + n * this->top_dim_, weight_diff);
					}
					// gradient w.r.t. bottom data, if necessary.
					if (propagate_down[i]) {
						this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
							bottom_diff + n * this->bottom_dim_);
					}
				}
			}
		}

		if (this->phase_ == TRAIN) {
			const int count = this->blobs_[0]->count();
			const int num = this->blobs_[0]->num();
			const int channels = this->blobs_[0]->channels();
			const int height = this->blobs_[0]->height();
			const int width = this->blobs_[0]->width();
			const int kernel_dim = this->blobs_[0]->count(1);
			// restore weight
			caffe_copy(count, w_buffer_.gpu_data(), this->blobs_[0]->mutable_gpu_data());
			// compute A grad
			caffe_gpu_mul(count, binary_w_.gpu_diff(), this->blobs_[0]->gpu_diff(), w_buffer_.mutable_gpu_diff());
			caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num,
				1, kernel_dim, Dtype(1. / kernel_dim), w_buffer_.gpu_diff(), multiplier_.gpu_data(),
				(Dtype)0., A_.mutable_gpu_diff());
			// compute meancenter grad
			if (height == 1 && width == 1) {
				caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num,
					1, channels, Dtype(1. / channels), this->blobs_[0]->gpu_diff(), multiplier_.gpu_data(),
					(Dtype)0., meancenter_.mutable_gpu_diff());
			}
			else {
				meancenter_backwark_kernel<Dtype> << <CAFFE_GET_BLOCKS(num*height*width), CAFFE_CUDA_NUM_THREADS >> >(
					num*height*width, num, channels, height, width,
					this->blobs_[0]->gpu_diff(), meancenter_.mutable_gpu_diff());
			}
			// compute w grad
			w_backwark_kernel<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
				count, num, channels, height, width,
				this->blobs_[0]->gpu_data(), binary_w_.gpu_diff(), this->blobs_[0]->gpu_diff(),
				A_.gpu_data(), A_.gpu_diff(), meancenter_.gpu_diff(), this->blobs_[0]->mutable_gpu_diff());
		}
	}

	template void BinaryConvolutionLayer<float>::binarizeGPUTo(Blob<float>* weights);
	template void BinaryConvolutionLayer<double>::binarizeGPUTo(Blob<double>* weights);

	INSTANTIATE_LAYER_GPU_FUNCS(BinaryConvolutionLayer);

}  // namespace caffe
