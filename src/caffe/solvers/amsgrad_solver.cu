#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"


namespace caffe {

template <typename Dtype>
__global__ void AMSGradUpdate(int N, Dtype* w, Dtype* g, Dtype* m, Dtype* v, Dtype* v_hat,
    Dtype beta1, Dtype beta2, Dtype eps_hat, Dtype corrected_local_rate, Dtype local_decay) {
  CUDA_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float mi = m[i] = m[i]*beta1 + gi*(1-beta1);
    float vi = v[i] = v[i]*beta2 + gi*gi*(1-beta2);
    v_hat[i] = max(v_hat[i], vi);
    g[i] = corrected_local_rate * mi / (sqrt(v_hat[i]) + eps_hat);// + local_decay*w[i];
  }
}
template <typename Dtype>
void amsgrad_update_gpu(int N, Dtype* w, Dtype* g, Dtype* m, Dtype* v, Dtype* v_hat, Dtype beta1,
    Dtype beta2, Dtype eps_hat, Dtype corrected_local_rate, Dtype local_decay) {
  AMSGradUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, w, g, m, v, v_hat, beta1, beta2, eps_hat, corrected_local_rate, local_decay);
  CUDA_POST_KERNEL_CHECK;
}
template void amsgrad_update_gpu<float>(int, float*, float*, float*, float*, float*,
    float, float, float, float, float);
template void amsgrad_update_gpu<double>(int, double*, double*, double*, double*, double*,
    double, double, double, double, double);

}  // namespace caffe
